#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h> 

hipEvent_t start, stop;     			
float elapsed_time_ms;       			

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;

	float Pvalue = 0;

	for (int k = 0; k < Width; ++k) {

		float Mdelement = Md[ty * Width + k];
		float Ndelement = Nd[k * Width + tx];
		Pvalue += (Mdelement * Ndelement);
	}

	Pd[ty * Width + tx] = Pvalue;
}

void MatrixMultiplication(float* M, float* N, float* P, int Width)
{
	int size = Width * Width * sizeof(float);
	float* Md, * Nd, * Pd;
	int k = 10;
	int l = 10;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)& Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	hipMalloc((void**)& Pd, size);

	dim3 dimBlock((k - 1) / Width + 1, (l - 1) / Width + 1);
	dim3 dimGrid(Width, Width);

	hipEventRecord(start, 0);			

	MatrixMulKernel << <dimGrid, dimBlock >> > (Md, Nd, Pd, Width);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     		
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);

	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
}

int main(void)
{

	void MatrixMultiplication(float*, float *, float*, int);

	const int Width = 10;
	float M[Width * Width], N[Width * Width], P[Width * Width];

	for (int i = 0; i < (Width * Width); i++) {
		M[i] = 2;
		N[i] = 1;
		P[i] = 0;
	}

	MatrixMultiplication(M, N, P, Width);


	for (int i = 0; i < (Width * Width); i++) {
		printf("%f \t", P[i]);
	}
	printf("\n");
	printf("�ð�: %f ms.", elapsed_time_ms); 

	return 0;
}
